#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/cpp_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void CppForwardKernel(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* img,
    const Dtype* ker,
    Dtype* top) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int c = (index / pixels_in_channel) % channels;
    const int n = (index / pixels_in_channel) / channels;
    // pretty much the same thing as with the CPU
    Dtype v = Dtype(0);
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        v += img[(n*channels + c)*pixels_in_channel + i_img*width + j_img] *
             ker[(n*kernel*kernel + i_ker*kernel + j_ker)*pixels_in_channel +
                 y*width + x];
      }
    }
    top[index] = v;
  }
}

template <typename Dtype>
void CPPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* img_data = bottom[0]->gpu_data();
  const Dtype* ker_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  // launch a cuda kernel for each of the output pixels
  const int count = bottom[0]->count();
  CppForwardKernel<Dtype>
      // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, batch_num_, channels_, height_, width_, kernel_, padding_,
      img_data, ker_data, top_data);
}

template <typename Dtype>
__global__ void CppBackwardKernel(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* img,
    const Dtype* src,
    Dtype* diff) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int n = (index / pixels_in_channel) / channels;
    // loop over each element of this filter
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        Dtype v = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          v += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        diff[(n*kernel*kernel + i_ker*kernel + j_ker)*pixels_in_channel +
             y*width + x] = v;
      }
    }
  }
}

template <typename Dtype>
void CPPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* src_data = top[0]->gpu_diff();
  Dtype* diff_data = bottom[1]->mutable_gpu_diff();
  const Dtype* img_data = bottom[0]->gpu_data();
  // launch a cuda kernel for each of the image pixels
  // will calculate gradients for each element of the corresponding filter
  const int count = bottom[0]->count();
  CppBackwardKernel<Dtype>
      // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, batch_num_, channels_, height_, width_, kernel_, padding_,
      img_data, src_data, diff_data);
}

INSTANTIATE_LAYER_GPU_FUNCS(CPPLayer);

}  // namespace caffe
