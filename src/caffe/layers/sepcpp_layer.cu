#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/sepcpp_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SepCppForwardKernel(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* img,
    const Dtype* ker,
    Dtype* top) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int c = (index / pixels_in_channel) % channels;
    const int n = (index / pixels_in_channel) / channels;
    // iterate over the pixel and its surroundings
    Dtype v = Dtype(0), t;
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      t = Dtype(0);
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        t += img[(n*channels + c)*pixels_in_channel + i_img*width + j_img] *
             ker[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x];
      }
      v += t * ker[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
                    y*width + x];
    }
    top[index] = v;
  }
}

template <typename Dtype>
void SepCPPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* img_data = bottom[0]->gpu_data();
  const Dtype* ker_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  SepCppForwardKernel<Dtype>
      // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, batch_num_, channels_, height_, width_, kernel_, padding_,
      img_data, ker_data, top_data);
}

template <typename Dtype>
__global__ void SepCppBackwardKernel_fast(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* src,
    const Dtype* img,
    const Dtype* ker,
    Dtype* diff) {
  // intermediate storage for vertical gradients
  // no VRAM impact as it is located in the low-level cache
  Dtype cache[SEPCPP_FAST_KERNEL_LIMIT];
  for (int i = 0; i < kernel; ++i) cache[i] = Dtype(0);
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int n = (index / pixels_in_channel) / channels;
    // both the horizontal and vertical gradients are accumulated within a
    // single pass; we iterate over each horizontal kernel element, calculating
    // its gradient and accumulating its contribution to each vertical kernel
    for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
        j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
      Dtype v = Dtype(0);
      // retrieve the horizontal kernel element for vertical gradient calc.
      Dtype h_j = ker[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x];
      for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
          i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
        Dtype t = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          t += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        // v accumulates gradient for this horizontal kernel element
        v += t * ker[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
                      y*width + x];
        // cache accumulates gradients for each vertical element, so we don't
        // have to run the entire second loop including channel accumulation
        cache[i_ker] += t * h_j;
      }
      diff[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x] = v;
    }
    // vertical gradient is already calculated and stored in the cache, so all
    // there's left to do is copy it into the blob
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      diff[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
            y*width + x] = cache[i_ker];
    }
  }
}

template <typename Dtype>
__global__ void SepCppBackwardKernel_any(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* src,
    const Dtype* img,
    const Dtype* ker,
    Dtype* diff) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int n = (index / pixels_in_channel) / channels;
    // horizontal gradient loop
    for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
        j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
      Dtype v = Dtype(0);
      for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
          i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
        Dtype t = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          t += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        v += t * ker[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
                      y*width + x];
      }
      diff[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x] = v;
    }
    // vertical gradient loop
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      Dtype v = Dtype(0);
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        Dtype t = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          t += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        v += t * ker[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x];
      }
      diff[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
            y*width + x] = v;
    }
  }
}

template <typename Dtype>
void SepCPPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* src_data = top[0]->gpu_diff();
  const Dtype* img_data = bottom[0]->gpu_data();
  const Dtype* ker_data = bottom[1]->gpu_data();
  Dtype* diff_data = bottom[1]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  // try to launch the optimized kernel, fall back to the default one
  if (kernel_ <= SEPCPP_FAST_KERNEL_LIMIT) {
    SepCppBackwardKernel_fast<Dtype>
        // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        (count, batch_num_, channels_, height_, width_, kernel_, padding_,
        src_data, img_data, ker_data, diff_data);
  } else {
    SepCppBackwardKernel_any<Dtype>
        // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
        (count, batch_num_, channels_, height_, width_, kernel_, padding_,
        src_data, img_data, ker_data, diff_data);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SepCPPLayer);

}  // namespace caffe
