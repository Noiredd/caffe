#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/sepcpp_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SepCppForwardKernel(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* img,
    const Dtype* ker,
    Dtype* top) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int c = (index / pixels_in_channel) % channels;
    const int n = (index / pixels_in_channel) / channels;
    // iterate over the pixel and its surroundings
    Dtype v = Dtype(0), t;
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      t = Dtype(0);
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        t += img[(n*channels + c)*pixels_in_channel + i_img*width + j_img] *
             ker[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x];
      }
      v += t * ker[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
                    y*width + x];
    }
    top[index] = v;
  }
}

template <typename Dtype>
void SepCPPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* img_data = bottom[0]->gpu_data();
  const Dtype* ker_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  SepCppForwardKernel<Dtype>
      // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, batch_num_, channels_, height_, width_, kernel_, padding_,
      img_data, ker_data, top_data);
}

template <typename Dtype>
__global__ void SepCppBackwardKernel(const int count,
    const int batch,
    const int channels,
    const int height,
    const int width,
    const int kernel,
    const int padding,
    const Dtype* src,
    const Dtype* img,
    const Dtype* ker,
    Dtype* diff) {
  CUDA_KERNEL_LOOP(index, count) {
    // find the current location given the index
    const int x = index % width;
    const int y = (index / width) % height;
    const int pixels_in_channel = height * width;
    const int n = (index / pixels_in_channel) / channels;
    // horizontal gradient loop
    for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
        j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
      Dtype v = Dtype(0);
      for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
          i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
        Dtype t = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          t += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        v += t * ker[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
                      y*width + x];
      }
      diff[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x] = v;
    }
    // vertical gradient loop
    for (int i_ker = max(padding-y, 0), i_img = y - padding + i_ker;
        i_ker < kernel && i_img < height; ++i_ker, ++i_img) {
      Dtype v = Dtype(0);
      for (int j_ker = max(padding-x, 0), j_img = x - padding + j_ker;
          j_ker < kernel && j_img < width; ++j_ker, ++j_img) {
        Dtype t = Dtype(0);
        for (int c = 0; c < channels; ++c) {
          int offset = (n*channels + c)*pixels_in_channel;
          t += src[offset + y*width + x] *
               img[offset + i_img*width + j_img];
        }
        v += t * ker[(n*2*kernel + j_ker)*pixels_in_channel + y*width + x];
      }
      diff[(n*2*kernel + kernel + i_ker)*pixels_in_channel +
            y*width + x] = v;
    }
  }
}

template <typename Dtype>
void SepCPPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* src_data = top[0]->gpu_diff();
  const Dtype* img_data = bottom[0]->gpu_data();
  const Dtype* ker_data = bottom[1]->gpu_data();
  Dtype* diff_data = bottom[1]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  SepCppBackwardKernel<Dtype>
      // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
      (count, batch_num_, channels_, height_, width_, kernel_, padding_,
      src_data, img_data, ker_data, diff_data);
}

INSTANTIATE_LAYER_GPU_FUNCS(SepCPPLayer);

}  // namespace caffe
